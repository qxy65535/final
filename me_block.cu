#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <math.h>
#include <hip/hip_runtime.h> 
//for __syncthreads()
#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif // !(__CUDACC_RTC__)
#include <hip/device_functions.h>

#include "c63.h"

__global__ void me_kernel(int padw, int padh, struct macroblock *mbs, int me_search_range, uint8_t *orig, uint8_t *ref, int cc)  
{
    __shared__ int sads[16*2][16*2];
    __shared__ int best_sad;
    sads[threadIdx.y][threadIdx.x] = 0;
    best_sad = INT_MAX;

    int mb_x = blockIdx.x;
    int mb_y = blockIdx.y;
    struct macroblock *mb = &mbs[mb_y * padw/8 + mb_x];

    int range = me_search_range;

    /* Half resolution for chroma channels. */
    if (cc > 0)
	    range /= 2;

    int left = mb_x*8 - range;
    int top = mb_y*8 - range;
    int right = mb_x*8 + range;
    int bottom = mb_y*8 + range;

    int w = padw;
    int h = padh;

    /* Make sure we are within bounds of reference frame */
    // TODO: Support partial frame bounds
    if (left < 0)
        left = 0;
    if (top < 0)
        top = 0;
    if (right > (w - 8))
        right = w - 8;
    if (bottom > (h - 8))
        bottom = h - 8;
    if (threadIdx.y >= (bottom-top) || threadIdx.x >= (right-left))
        return;

    int mx = mb_x * 8;
    int my = mb_y * 8;

    int row = blockIdx.y*8;
    int col = blockIdx.x*8;

    int i,j;
    for (i=0; i<8; ++i)
    {
        for (j=0; j<8; ++j)
        {
            int result = abs(*(orig+(row+i)*w+col+j) - *(ref+(top+threadIdx.y+i)*w+left+threadIdx.x+j));
            atomicAdd(&sads[threadIdx.y][threadIdx.x], result);
        }
    }

    // 找出小的sad值
    atomicMin(&best_sad, sads[threadIdx.y][threadIdx.x]);
    __syncthreads();

    // 找出最相似的参考块
    for (i=0; i<(bottom-top); ++i)
    {
        for (j=0; j<(right-left); ++j)
        {
            if (sads[i][j] == best_sad)
            {
                mb->mv_x = left + j - mx;
                mb->mv_y = top + i - my;
                i = bottom-top;
                break;
            }
        }
    }
    mb->use_mv = 1;
}  

extern "C" void me_block_cuda(struct c63_common *cm, uint8_t *orig_host, uint8_t *ref_host, int cc)
{
    struct macroblock *mbs;
    uint8_t *orig, *ref;

    int size_mbs = 0;
    int size_orig = 0;
    int size_ref = 0;
    switch (cc)
    {
    case 0:
        size_mbs = cm->mb_rows * cm->mb_cols * sizeof(struct macroblock);
        size_orig = cm->width * cm->height;
        size_ref = cm->ypw * cm->yph;
        break;
    case 1:
        size_mbs = cm->mb_rows/2 * cm->mb_cols/2 * sizeof(struct macroblock);
        size_orig = cm->width * cm->height;
        size_ref = cm->upw * cm->uph;
        break;
    case 2:
        size_mbs = cm->mb_rows/2 * cm->mb_cols/2 * sizeof(struct macroblock);
        size_orig = cm->width * cm->height;
        size_ref = cm->vpw * cm->vph;
        break;
    }
    hipMalloc((void **)&mbs, size_mbs);
    hipMalloc((void **) &orig, size_orig);
    hipMalloc((void **) &ref, size_ref);
    // hipMemcpy(mbs, cm->curframe->mbs[cc], size_mbs, hipMemcpyHostToDevice);
    hipMemcpy(orig, orig_host, size_orig, hipMemcpyHostToDevice);
    hipMemcpy(ref, ref_host, size_ref, hipMemcpyHostToDevice);

    int grid_x = cc>0 ? cm->mb_cols/2:cm->mb_cols;
    int grid_y = cc>0 ? cm->mb_rows/2:cm->mb_rows;
    dim3 dimGrid(grid_x, grid_y);
    dim3 dimBlock(cm->me_search_range*2,cm->me_search_range*2);

    me_kernel<<<dimGrid, dimBlock>>>(cm->padw[cc], cm->padh[cc], mbs, 
                                    cm->me_search_range, orig, ref, cc);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    hipMemcpy(cm->curframe->mbs[cc], mbs, size_mbs, hipMemcpyDeviceToHost);

    hipFree(mbs);
    hipFree(orig);
    hipFree(ref);
}